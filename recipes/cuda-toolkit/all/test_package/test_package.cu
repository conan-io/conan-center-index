
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>

int main(void)
{
    int devs = 0;
    hipError_t status = hipGetDeviceCount(&devs);
    printf("devs available: %d (status %d: \"%s\")\n", devs, status, hipGetErrorString(status));
    return EXIT_SUCCESS;
}