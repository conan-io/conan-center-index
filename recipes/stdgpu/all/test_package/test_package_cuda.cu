#include "hip/hip_runtime.h"
// Based on https://github.com/stotko/stdgpu/blob/32e0517/examples/cuda/vector.cu
#include <thrust/copy.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>

#include <stdgpu/iterator.h> // device_begin, device_end
#include <stdgpu/memory.h>   // createDeviceArray, destroyDeviceArray
#include <stdgpu/platform.h> // STDGPU_HOST_DEVICE
#include <stdgpu/vector.cuh> // stdgpu::vector

#include <cstdlib>
#include <iostream>

__global__ void insert_neighbors_with_duplicates(const int *d_input, const stdgpu::index_t n,
                                                 stdgpu::vector<int> vec) {
    stdgpu::index_t i = static_cast<stdgpu::index_t>(blockIdx.x * blockDim.x + threadIdx.x);

    if (i >= n)
        return;

    int num = d_input[i];
    int num_neighborhood[3] = {num - 1, num, num + 1};

    for (int num_neighbor : num_neighborhood) {
        vec.push_back(num_neighbor);
    }
}

int sum_stdgpu(stdgpu::index_t n) {
    int *d_input = createDeviceArray<int>(n);
    auto vec = stdgpu::vector<int>::createDeviceObject(3 * n);

    thrust::sequence(stdgpu::device_begin(d_input), stdgpu::device_end(d_input), 1);

    stdgpu::index_t threads = 32;
    stdgpu::index_t blocks = (n + threads - 1) / threads;
    insert_neighbors_with_duplicates<<<static_cast<unsigned int>(blocks),
                                       static_cast<unsigned int>(threads)>>>(d_input, n, vec);
    hipDeviceSynchronize();

    auto range_vec = vec.device_range();
    int sum = thrust::reduce(range_vec.begin(), range_vec.end(), 0, thrust::plus<int>());

    destroyDeviceArray<int>(d_input);
    stdgpu::vector<int>::destroyDeviceObject(vec);

    return sum;
}

int main() {
    const int n = 20;
    const int sum_closed_form = 3 * (n * (n + 1) / 2);
    std::cout << "Sum: " << sum_stdgpu(n) << ", expected: " << sum_closed_form << std::endl;
    return EXIT_SUCCESS;
}
